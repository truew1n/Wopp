#include <iostream>
#include <windows.h>

#include "AudioController.hpp"


uint32_t CalcAOB(uint32_t Value, uint32_t AOT)
{
    return (Value / AOT) + ((Value % AOT) > 0);
}

LRESULT CALLBACK WinProcedure(HWND HWnd, UINT UMsg, WPARAM WParam, LPARAM LParam);

int main(void)
{
    AudioController Controller = AudioController();
    Controller.Add(L"Assets\\Audio\\Napoleon's song (Amour Plastique- slowed version).wav");
    Controller.Add(L"Assets\\Audio\\Soviet  March.wav");
    Controller.Add(L"Assets\\Audio\\Aiyo - Below Freezing.wav");
    Controller.Add(L"Assets\\Audio\\George Michael - Careless Whisper (Lyrics).wav");
    Controller.Add(L"Assets\\Audio\\Selfless.wav");
    Controller.Add(L"Assets\\Audio\\Horse Race.wav");
    
    Controller.Start();

    HINSTANCE WinInstance = GetModuleHandleW(NULL);
    
    WNDCLASSW WinClass = {0};
    WinClass.lpszClassName = L"Wopp";
    WinClass.hbrBackground = (HBRUSH) COLOR_WINDOW;
    WinClass.hCursor = LoadCursor(NULL, IDC_ARROW);
    WinClass.hInstance = WinInstance;
    WinClass.lpfnWndProc = WinProcedure;

    if(!RegisterClassW(&WinClass)) return -1;

    uint32_t Width = 800;
    uint32_t Height = 600;

    RECT WindowRect = { 0 };
    WindowRect.right = Width;
    WindowRect.bottom = Height;
    WindowRect.left = 0;
    WindowRect.top = 0;

    AdjustWindowRect(&WindowRect, WS_OVERLAPPEDWINDOW | WS_VISIBLE, 0);
    HWND Window = CreateWindowW(
        WinClass.lpszClassName,
        L"Wopp - Wave Audio Player",
        WS_OVERLAPPEDWINDOW | WS_VISIBLE,
        CW_USEDEFAULT, CW_USEDEFAULT,
        WindowRect.right - WindowRect.left,
        WindowRect.bottom - WindowRect.top,
        NULL, NULL,
        NULL, NULL
    );
    
    GetWindowRect(Window, &WindowRect);

    uint32_t BitmapWidth = Width;
    uint32_t BitmapHeight = Height;

    uint32_t BytesPerPixel = 4;

    uint32_t BitmapTotalSize = BitmapWidth * BitmapHeight;
    uint32_t DisplayTotalSize = BitmapTotalSize * BytesPerPixel;

    void *Display;
    hipMallocManaged(&Display, DisplayTotalSize);

    uint32_t AOT = 1024;
    uint32_t DisplayAOB = CalcAOB(BitmapTotalSize, AOT);

    BITMAPINFO BitmapInfo;
    BitmapInfo.bmiHeader.biSize = sizeof(BitmapInfo.bmiHeader);
    BitmapInfo.bmiHeader.biWidth = BitmapWidth;
    BitmapInfo.bmiHeader.biHeight = -BitmapHeight;
    BitmapInfo.bmiHeader.biPlanes = 1;
    BitmapInfo.bmiHeader.biBitCount = 32;
    BitmapInfo.bmiHeader.biCompression = BI_RGB;

    HDC hdc = GetDC(Window);

    MSG msg = { 0 };
    int32_t running = 1;
    while (running) {

        while (PeekMessageW(&msg, NULL, 0, 0, PM_REMOVE)) {
            switch (msg.message) {
                case WM_QUIT: {
                    running = 0;
                    break;
                }
            }
            TranslateMessage(&msg);
            DispatchMessageW(&msg);
        }
        
        StretchDIBits(
            hdc, 0, 0,
            BitmapWidth, BitmapHeight,
            0, 0,
            BitmapWidth, BitmapHeight,
            Display, &BitmapInfo,
            DIB_RGB_COLORS,
            SRCCOPY
        );
    }

    hipFree(Display);

    Controller.Free();
    return 0;
}

LRESULT CALLBACK WinProcedure(HWND HWnd, UINT UMsg, WPARAM WParam, LPARAM LParam)
{
    switch (UMsg) {
        case WM_DESTROY: {
            PostQuitMessage(0);
            break;
        }
        default: {
            return DefWindowProcW(HWnd, UMsg, WParam, LParam);
            break;
        }
    }
    return 0;
}