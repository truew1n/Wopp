#include <iostream>
#include <windows.h>

#include "AudioController.hpp"


uint32_t CalcAOB(uint32_t Value, uint32_t AOT)
{
    return (Value / AOT) + ((Value % AOT) > 0);
}

LRESULT CALLBACK WinProcedure(HWND HWnd, UINT UMsg, WPARAM WParam, LPARAM LParam);

int main(void)
{
    AudioController Controller = AudioController();

    WIN32_FIND_DATAW findFileData;
    HANDLE hFind;

    LPCWSTR searchPath = L"Assets\\Audio\\*.wav";
    hFind = FindFirstFileW(searchPath, &findFileData);
    if (hFind == INVALID_HANDLE_VALUE) {
        std::wcerr << L"Error finding files in directory" << std::endl;
        return 1;
    }

    do {
        std::wcout << findFileData.cFileName << std::endl;
        Controller.Add(L"Assets\\Audio\\" + std::wstring(findFileData.cFileName));
    } while (FindNextFileW(hFind, &findFileData) != 0);
    FindClose(hFind);

    // Controller.Add(L"Assets\\Audio\\Alle Farben feat. YouNotUs- Please Tell Rosie [Official Video].wav");
    // Controller.Add(L"Assets\\Audio\\Kungs vs Cookin on 3 Burners - This Girl (Official Music Video).wav");
    // Controller.Add(L"Assets\\Audio\\SAIL - AWOLNATION (Unofficial Video).wav");

    Controller.Start();

    ShowWindow(GetConsoleWindow(), SW_HIDE);
    HINSTANCE WinInstance = GetModuleHandleW(NULL);
    
    WNDCLASSW WinClass = {0};
    WinClass.lpszClassName = L"Wopp";
    WinClass.hbrBackground = (HBRUSH) COLOR_WINDOW;
    WinClass.hCursor = LoadCursor(NULL, IDC_ARROW);
    WinClass.hInstance = WinInstance;
    WinClass.lpfnWndProc = WinProcedure;

    if(!RegisterClassW(&WinClass)) return -1;

    uint32_t Width = 800;
    uint32_t Height = 600;

    RECT WindowRect = { 0 };
    WindowRect.right = Width;
    WindowRect.bottom = Height;
    WindowRect.left = 0;
    WindowRect.top = 0;

    AdjustWindowRect(&WindowRect, WS_OVERLAPPEDWINDOW | WS_VISIBLE, 0);
    HWND Window = CreateWindowW(
        WinClass.lpszClassName,
        L"Wopp - Wave Audio Player",
        WS_OVERLAPPEDWINDOW | WS_VISIBLE,
        CW_USEDEFAULT, CW_USEDEFAULT,
        WindowRect.right - WindowRect.left,
        WindowRect.bottom - WindowRect.top,
        NULL, NULL,
        NULL, NULL
    );
    
    GetWindowRect(Window, &WindowRect);

    uint32_t BitmapWidth = Width;
    uint32_t BitmapHeight = Height;

    uint32_t BytesPerPixel = 4;

    uint32_t BitmapTotalSize = BitmapWidth * BitmapHeight;
    uint32_t DisplayTotalSize = BitmapTotalSize * BytesPerPixel;

    void *Display;
    hipMallocManaged(&Display, DisplayTotalSize);

    uint32_t AOT = 1024;
    uint32_t DisplayAOB = CalcAOB(BitmapTotalSize, AOT);

    BITMAPINFO BitmapInfo;
    BitmapInfo.bmiHeader.biSize = sizeof(BitmapInfo.bmiHeader);
    BitmapInfo.bmiHeader.biWidth = BitmapWidth;
    BitmapInfo.bmiHeader.biHeight = -BitmapHeight;
    BitmapInfo.bmiHeader.biPlanes = 1;
    BitmapInfo.bmiHeader.biBitCount = 32;
    BitmapInfo.bmiHeader.biCompression = BI_RGB;

    HDC hdc = GetDC(Window);

    MSG msg = { 0 };
    int32_t running = 1;
    while (running) {

        while (PeekMessageW(&msg, NULL, 0, 0, PM_REMOVE)) {
            switch (msg.message) {
                case WM_QUIT: {
                    running = 0;
                    break;
                }
            }
            TranslateMessage(&msg);
            DispatchMessageW(&msg);
        }
        
        StretchDIBits(
            hdc, 0, 0,
            BitmapWidth, BitmapHeight,
            0, 0,
            BitmapWidth, BitmapHeight,
            Display, &BitmapInfo,
            DIB_RGB_COLORS,
            SRCCOPY
        );
    }

    hipFree(Display);

    Controller.Free();
    return 0;
}

LRESULT CALLBACK WinProcedure(HWND HWnd, UINT UMsg, WPARAM WParam, LPARAM LParam)
{
    switch (UMsg) {
        case WM_DESTROY: {
            PostQuitMessage(0);
            break;
        }
        default: {
            return DefWindowProcW(HWnd, UMsg, WParam, LParam);
            break;
        }
    }
    return 0;
}